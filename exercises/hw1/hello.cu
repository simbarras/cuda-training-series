
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 3
#define M 4

__global__ void hello(){

  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main(){
  hello<<<N, M>>>();
  hipDeviceSynchronize();
}

